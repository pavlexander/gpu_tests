#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include "hip/hip_runtime.h"
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h>

//#include <cassert>
//#include <cstdio>
//#include <cfloat>
//#include <cinttypes>
//#include <algorithm>
//#include <memory>
//#include <hiprand/hiprand_kernel.h>

//#include "float.h"
#include <stdio.h>

#include "kernel.h"
#include <iostream>

//#define _SIZE_T_DEFINED

// https://stackoverflow.com/questions/6061565/setting-up-visual-studio-intellisense-for-cuda-kernel-calls
// nvcc does not seem to like variadic macros, so we have to define
// one for each kernel parameter list:
#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define CUDA_CALLABLE_MEMBER
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

// Now launch your kernel using the appropriate macro:
//kernel KERNEL_ARGS2(dim3(nBlockCount), dim3(nThreadCount)) (param1);

/*
#ifndef __HIPCC__
#define __HIPCC__
#endif
*/

/*
#ifndef __cplusplus
#define __cplusplus
#endif
*/

// Texture reference
//texture<float2, 2> texref;

// restrict
// https://devblogs.nvidia.com/cuda-pro-tip-optimize-pointer-aliasing/
// http://www.orangeowlsolutions.com/archives/310

// shared
// https://stackoverflow.com/questions/16754885/is-it-worthwhile-to-pass-kernel-parameters-via-shared-memory
// https://stackoverflow.com/questions/7903566/how-is-2d-shared-memory-arranged-in-cuda?rq=1
// https://devblogs.nvidia.com/using-shared-memory-cuda-cc/

// shared and blocks
// https://stackoverflow.com/questions/43195914/gpu-shared-memory-practical-example

// blocks !!
// https://algoslaves.wordpress.com/2013/09/16/real-life-cuda-example-time-series-denoising-with-daubechies-4-discrete-wavelet-transform-with-managedcuda-and-c/


// has to be extern, so that managed cuda could see it
// todo: try without __restrict__ 
// todo: try using shared memory for parameters vs passing as parameters
// todo: test what |extern "C"| does exactly with generated code
// todo: check what blocks and grids are
// todo: use tenary operator
// todo: use 1D array instead of 2D
extern "C"
{
	//__device__ __constant__ int width;
	//__device__ __constant__ int inputCount;
	//__device__ __constant__ int height;

	//kernel code
	__global__ void proccess(
		unsigned char* __restrict__ output,
		double* __restrict__ outputCalc,

		const int* __restrict__ in1,
		const int* __restrict__ in2,
		const double* __restrict__ in3,
		const unsigned char* __restrict__  in4, // unsigned char

		const int inputCount,
		const int width,
		const int height
	)
	{
		//int index = threadIdx.x;
		int index = blockIdx.x * blockDim.x + threadIdx.x;

		if (index >= inputCount) {
			return;
		}
		
		bool isTrue = false;
		int varA = in1[index];
		int varB = in2[index];

		// __shared__ double calculatable = 0;;
		double calculatable = 0;
		//bool result = false;

		// __syncthreads();

		// https://stackoverflow.com/questions/8011376/when-is-cudas-shared-memory-useful
		// __shared__ float in3_shared[sizeof(in3)];
		// __shared__ float in4_shared[sizeof(in4)][];
		bool isLastFirstCondition = false;
		for (int row = 0; row < height; row++)
		{
			// in3_shared[index] = in3[index];
			// __syncthreads();

			if (isTrue)
			{
				int idx = width * row + varA;

				if (!in4[idx]) {
					continue;
				}

				calculatable = calculatable + in3[row];
				isTrue = false;

				isLastFirstCondition = true;
			}
			else
			{
				int idx = width * row + varB;

				if (!in4[idx]) {
					continue;
				}

				calculatable = calculatable - in3[row];
				isTrue = true;

				isLastFirstCondition = false;
			}
		}

		/*
		outputCalc[0] = in4[0];
		outputCalc[1] = in4[1];
		outputCalc[2] = in4[2];
		outputCalc[3] = in4[3];
		outputCalc[4] = 1111;*/

		output[index] = isLastFirstCondition;
		outputCalc[index] = calculatable;

		/*
		// testing
		output[index] = in4[0];
		outputCalc[index] = index;*/
	}
}

__global__ void proccess2(
	bool* __restrict__ output,
	double* __restrict__ outputCalc
)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	// testing
	output[index] = true;
	outputCalc[index] = 15.4;
}

int CudaProccess(
	unsigned char* output,
	const size_t output_size, // unsigned int
	double* outputCalc,
	const size_t outputCalc_size,

	const int* in1,
	const size_t in1_size,
	const int* in2,
	const size_t in2_size,
	const double* in3,
	const size_t in3_size,
	const unsigned char* in4,
	const size_t in4_size,

	const int inputCount,
	const int width,
	const int height

	//unsigned char* __restrict__ output,
	//const int* __restrict__ in1,
	//const int* __restrict__ in2,
	//const double* __restrict__ in3,
	//const unsigned char* __restrict__  in4
) {
	//
	// Create int arrays on the CPU.
	// ('h' stands for "host".)
	//
	// SKIP (see method's input params)

	//
	// Create corresponding int arrays on the GPU.
	// ('d' stands for "device".)
	//
	// init dev variables
	unsigned char* d_output;
	double* d_outputCalc;
	int* d_in1;
	int* d_in2;
	double* d_in3;
	unsigned char* d_in4;
	//int d_inputCount;
	//int d_height;
	//int d_width;

	// calculate native total sizes
	//int constSize = sizeof(int);
	unsigned int output_totalSize = output_size * sizeof(unsigned char);
	unsigned int outputCalc_totalSize = outputCalc_size * sizeof(double);
	unsigned int in1_totalSize = in1_size * sizeof(int);
	unsigned int in2_totalSize = in2_size * sizeof(int);
	unsigned int in3_totalSize = in3_size * sizeof(double); // TBD: issue?
	unsigned int in4_totalSize = in4_size * sizeof(unsigned char);

	// allocate memory for device variables
	hipMalloc(&d_output, output_totalSize);
	hipMalloc(&d_outputCalc, outputCalc_totalSize);
	hipMalloc(&d_in1, in1_totalSize);
	hipMalloc(&d_in2, in2_totalSize);
	hipMalloc(&d_in3, in3_totalSize);
	hipMalloc(&d_in4, in4_totalSize);
	
	// Allocate Unified Memory � accessible from CPU or GPU
	//hipMallocManaged(&output, output_totalSize);
	//hipMallocManaged(&outputCalc, outputCalc_totalSize);

	// write host -> device
	hipMemcpy(d_in1, in1, in1_totalSize, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, in2_totalSize, hipMemcpyHostToDevice);
	hipMemcpy(d_in3, in3, in3_totalSize, hipMemcpyHostToDevice);
	hipMemcpy(d_in4, in4, in4_totalSize, hipMemcpyHostToDevice);

	//hipMalloc((void**)&d_inputCount, constSize);
	//hipMalloc((void**)&d_height, constSize);
	//hipMalloc((void**)&d_width, constSize);

	/*
	std::string s = " ";
	std::cout << "Before" << std::endl;
	std::cout << in4[0] << std::endl;
	for (size_t i = 0; i < 10; i++)
	{
		//std::cout << i << s << output[i] << s << outputCalc[i] << std::endl;
	}*/

	// execute
	//dim3 dimBlock(inputCount, 1);
	//dim3 dimGrid(1, 1);
	//proccess KERNEL_ARGS2(dimGrid, dimBlock) (d_output, d_outputCalc, d_in1, d_in2, d_in3, d_in4, inputCount, width, height);
	//proccess KERNEL_ARGS2(inputCount, 1) (d_output, d_outputCalc, d_in1, d_in2, d_in3, d_in4, inputCount, width, height);
	//proccess << <inputCount, 1 >> > (d_output, d_outputCalc, d_in1, d_in2, d_in3, d_in4, inputCount, width, height);

	//proccess2 KERNEL_ARGS2(40, 1) (d_output, d_outputCalc);
	// 
	// Attempt 2: (working)
	//int blockDimensions = 256;
	//int gridDimensions = (inputCount + blockDimensions - 1) / blockDimensions;
	//proccess << <gridDimensions, blockDimensions >>> (d_output, d_outputCalc, d_in1, d_in2, d_in3, d_in4, inputCount, width, height);

	
	// Attempt 3: (working)
	int blockSize;      // The launch configurator returned block size 
	int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
	int gridSize;       // The actual grid size needed, based on input size 
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, proccess, 0, inputCount);
	//int blockSize = 1024;
	//int minGridSize = 56;
	//int gridSize  = 1024;
	proccess KERNEL_ARGS2(gridSize, blockSize) (d_output, d_outputCalc, d_in1, d_in2, d_in3, d_in4, inputCount, width, height);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	//
	// Copy output array from GPU back to CPU.
	//
	hipMemcpy(output, d_output, output_totalSize, hipMemcpyDeviceToHost);
	hipMemcpy(outputCalc, d_outputCalc, outputCalc_totalSize, hipMemcpyDeviceToHost);

	/*
	std::cout << "After" << std::endl;
	for (size_t i = 0; i < 10; i++)
	{
		std::cout << i << s << output[i] << s << outputCalc[i] << std::endl;
	}*/

	//
	// Free up the arrays on the GPU.
	//
	hipFree(d_output);
	hipFree(d_outputCalc);
	hipFree(d_in1);
	hipFree(d_in2);
	hipFree(d_in3);
	hipFree(d_in4);

	// check for error
	// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error name: %s\n", hipGetErrorName(error));
		printf("CUDA error description: %s\n", hipGetErrorString(error));
		//exit(-1);
		return -1;
	}

	return 0;
}